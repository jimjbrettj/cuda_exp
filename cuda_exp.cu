 
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/*
Common workflow of cuda programs:
    1) Allocate host memory and initialized host data
    2) Allocate device memory
    3) Transfer input data from host to device memory
    4) Execute kernels
    5) Transfer output from device memory to host
*/

__global__ void add(int a, int b, int *c){
    *c = a + b;
}

__global__ void test(int a, int *c){
    *c = a;
}

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main(){
    // Allocate host memory and initialized host data
    int a, b, c;
    int *dev_c;
    a = 3;
    b = 4;
    // Allocate device memory
    gpuErrchk(hipMalloc((void**) &dev_c, sizeof(int)));
    // Execute kernels
    cuda_hello<<<1,1>>>();

    hipDeviceSynchronize();
    //test<<<1,1>>>(a, dev_c);
    add<<<1,1>>>(a, b, dev_c);
    // Transfer output from device memory to host
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d + %d = %d\n", a, b, c);
    gpuErrchk( hipPeekAtLastError() );
    // Free device memory
    hipFree(dev_c);
    return 0;
}

// #define N 10

// __global__ void vector_add(float *out, float *a, float *b, int n) {
//     for(int i = 0; i < n; i++){
//         out[i] = a[i] + b[i];
//     }
// }

// int main(){
//     float *a, *b, *out; 

//     // Allocate memory
//     a   = (float*)malloc(sizeof(float) * N);
//     b   = (float*)malloc(sizeof(float) * N);
//     out = (float*)malloc(sizeof(float) * N);

//     // Initialize array
//     for(int i = 0; i < N; i++){
//         a[i] = 1.0f; b[i] = 2.0f;
//     }

//     // Main function
//     //vector_add(out, a, b, N);
//     vector_add<<<1,1>>>(out, a, b, N);

//     for(int i = 0; i < N; i++){
//         printf("%f ", out[i]);
//     }
//     printf("\n");
//    	return 0;
// }
